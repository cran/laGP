#include "hip/hip_runtime.h"
// module load cuda
// nvcc -arch=sm_20 -c -Xcompiler -fPIC alc_gpu.cu -o alc_gpu.o


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern "C" {
  #include "alc_gpu.h"
}

// #define SUM2

// #define TIMINGS
#ifdef TIMINGS
#include <sys/time.h>
#endif

#define SDEPS sqrt(2.220446e-16)

/*
 * check_gpu_error:
 *
 * checking for errors after kernel calls 
 */

static void check_gpu_error(const char *msg) {
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err)
        printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err));
}

/*
 * NearestPowerOf2:
 *
 * finds the nearest power of 2 greater than n
 */

__device__ int NearestPowerOf2 (int n)
  {
    if (!n) return n;  //(0 == 2^0)
 
    int x = 1;
    while(x < n) { x <<= 1; }
  	
    return x;
}


/*
 * alc_kernel:
 *
 * return s2' component of the ALC calculation of the
 * expected reduction in variance calculation at locations 
 * Xcand averaging over reference locations Xref: 
 * ds2 = s2 - s2', where the s2s are at Xref and the
 * s2' incorporates Xcand, and everything is averaged
 * over Xref.
 */

__global__ void alc_kernel(const int n, const int m, double *X, double *Ki, 
         const double d, const double g, const double phi, const int ncand, 
         double *Xcand, const int nref, double *Xref, double *k, double *alc)
{
  unsigned tid, tid2, halfPoint, bid, bidm, nTotalThreads;
  double mui, kx_tid, gvec_tid, /* dot, ktGmuik, */ ktKikx_sum;

  extern __shared__ double s[];
  double* gvec = s; /* size n */
  double* kxy = &s[n];  /* size nref */
  double* kx = &s[n+nref];  /* size n */
  double* kx_gvec = &s[2*n+nref]; /* size n */
  double* Xc = &s[3*n+nref]; /* size m */
  double* k_g = kx; // &s[3*n+2*nref+m]; /* size n */
  double* ktGmui_k = kx_gvec; // &s[4*n+2*nref+m]; /* size n */

  /* thread indices stored in registers */
  tid = threadIdx.x; 
  bid = blockIdx.x;
  bidm = bid*m;

  /* copy row of Xcand into faster shared memory */
  if(tid < m) Xc[tid] = Xcand[bidm + tid];
  __syncthreads(); 
 
  /* calculate covariance between candidate and tid-th data point */
  /* each thread accesses global memory m times; might make sense to copy 
     like with Ki */
  kx_tid = 0.0;
  for(unsigned int k=0; k<m; k++)
     kx_tid += (Xc[k] - X[tid*m+k])*(Xc[k] - X[tid*m+k]);
  kx[tid] = exp(0.0 - kx_tid/d);
  __syncthreads();

  /* gvec calculation and dot product preparation */
  gvec_tid = 0;
  for(unsigned int i=0; i<n; i++)	gvec_tid += kx[i] * Ki[i*n+tid];
  kx_gvec[tid] = gvec_tid * kx[tid];
  __syncthreads();

  /* kx is no longer needed as of this point in the program;
     will later be used by k_g  */

  /* reduction for dot product */
  nTotalThreads = NearestPowerOf2(blockDim.x);
  while(nTotalThreads > 1) {
    halfPoint = (nTotalThreads >> 1);
    if(tid < halfPoint) {
      tid2 = tid + halfPoint;
      if(tid2 < blockDim.x) kx_gvec[tid] += kx_gvec[tid2];
    }
    __syncthreads();
    nTotalThreads = halfPoint;
  } 

  /* mui <- drop(1 + Zt$g - t(kx) %*% Kikx) */
  mui = 1.0 + g - kx_gvec[0]; 
  /* finish gvec calculation */
  gvec[tid] = 0.0 - gvec_tid/mui;
  // ktKikx_sum = 0.0;
  /* no need to sync threads */

  /* kx_gvec is no longer needed as of this point in the program;
     will later be used by ktGmui_k  */

  /* preparation of kxy: could thread for nref */
  if(tid == 0) {
     for(unsigned int i=0; i<nref; i++) {
       kx_tid = 0.0;  // re-using kx_tid from above 
       for(unsigned int k=0; k<m; k++)
          kx_tid += (Xc[k] - Xref[i*m+k])*(Xc[k] - Xref[i*m+k]);
       kxy[tid + i] = exp(0.0 - kx_tid/d);
     }
  }

  /* preparation of alc */
  if(tid == 1) alc[bid] = 0.0;
   __syncthreads();

 
  /* skip if numerical problems */
  if(mui > SDEPS) {
         
    /* use g, mu, and kxy to calculate ktKik.x */
    /* loop over all of the nref reference locations: 
       when nref is bigger we might want to thread this */
    for(unsigned int r=0; r<nref; r++) {

      /* ktGmui = t(k) %*% Gmui %*% k */
      double ktGmui_tid = 0;
      for(unsigned int j=0; j<n; j++)
        ktGmui_tid += k[r*n+j] * gvec[tid]*gvec[j]*mui;
      ktGmui_k[tid] = ktGmui_tid * k[r*n+tid];
      k_g[tid] = k[r*n+tid]*gvec[tid];
      __syncthreads();

      /* two reductions for ktGimuk and kg (re-using dot)  */
      nTotalThreads = NearestPowerOf2(blockDim.x);
#ifdef SUM2
      /* Option 1: each active thread does double work */
      while(nTotalThreads > 1) {
         halfPoint = (nTotalThreads >> 1);
         if(tid < halfPoint) {
           tid2 = tid + halfPoint;
           if(tid2 < blockDim.x) {
                 ktGmui_k[tid] += ktGmui_k[tid2];
                 k_g[tid] += k_g[tid2];
           }
         }
         __syncthreads();
         nTotalThreads = halfPoint;
       }
#else
       /* Option 2: the > n/2 idle threads pick up the second sum */
       /* first the threads to double-duty until power of 2 */
       halfPoint = (nTotalThreads >> 1);
       if(tid < halfPoint) {
          tid2 = tid + halfPoint;
          if(tid2 < blockDim.x) {
            ktGmui_k[tid] += ktGmui_k[tid2];
            k_g[tid] += k_g[tid2];
          }
        }
        __syncthreads();

        /* now its a power of to so we can do 2 parallel reductions */
        if(tid < halfPoint){
          if(tid < halfPoint/2) { // The first half of the threads work on ktGmui_k
            for(unsigned int s=halfPoint/2; s>0; s>>=1) {
              if(tid < s) ktGmui_k[tid] += ktGmui_k[tid + s];
              __syncthreads();
            } 
          } else { // The second half of the threads works on k_g
            tid = tid - (halfPoint/2);
            for(unsigned int s=halfPoint/2; s>0; s>>=1) {
              if (tid < s) k_g[tid] += k_g[tid + s];
              __syncthreads();
            }
          }
        }
#endif

      /* finish ktKikx calculation */
      if(tid == 0) ktKikx_sum /*+*/= ktGmui_k[0] + 2.0*k_g[0]*kxy[r] + kxy[r]*kxy[r]/mui;
      // __syncthreads();
    }
    
    /* calculate the ALC */
    /* when nref is bigger we might want to thread-reduce this */
    if(tid == 0) alc[bid] = phi*ktKikx_sum/((n-2.0)*((double) nref));
  }
}


extern "C" {

/*
 * num_gpus:
 *
 * a wrapper function to check how many gpus there are
 */

int num_gpus(void)  
  {
    int count;
    hipError_t success = hipGetDeviceCount(&count);
    if(success == hipSuccess) return count;
    else return 0;
  }

/*
 * alcGP_gpu:
 *
 * calculate ALC stats on a GPU; for C-version and more comments 
 * see alcGP in gp.c 
 */

void alc_gpu(double d, double g, double phi, int m, int n, double *X,
             double *Ki, int ncand, double *Xcand, int nref, double *Xref, 
             double *k, double *alcv, int gpu)
  {
    double *d_k, *d_X, *d_Ki, *d_Xref, *d_Xcand, *d_alcv;

#ifdef TIMINGS
    struct timeval t1, t2;

    /* for timing memory copies */
    gettimeofday(&t1, 0);
#endif

    hipError_t devsuccess = hipSetDevice(gpu);
    assert(devsuccess == hipSuccess);

    /* copy to GPU */
    hipMalloc((void**) &d_X, (n*m) * sizeof(double));
    hipMemcpy(d_X, X, (n*m) * sizeof(double), hipMemcpyHostToDevice);
    check_gpu_error("X copy");
    hipMalloc((void**) &d_Ki, (n*n) * sizeof(double));
    hipMemcpy(d_Ki, Ki, (n*n) * sizeof(double), hipMemcpyHostToDevice);
    check_gpu_error("Ki copy");
    hipMalloc((void**) &d_Xref, (nref*m) * sizeof(double));
    hipMemcpy(d_Xref, Xref, (nref*m) * sizeof(double), hipMemcpyHostToDevice);
    check_gpu_error("Xref copy");
    hipMalloc((void**) &d_Xcand, (ncand*m) * sizeof(double));
    hipMemcpy(d_Xcand, Xcand, (ncand*m) * sizeof(double), hipMemcpyHostToDevice);
    check_gpu_error("Xcand copy");
    hipMalloc((void**) &d_k, (nref*n) * sizeof(double));
    hipMemcpy(d_k, k, (nref*n) * sizeof(double), hipMemcpyHostToDevice);
    check_gpu_error("k copy");
    /* allocate output on GPU */
    hipMalloc((void**) &d_alcv, ncand * sizeof(double));
    // hipMemset((void**) &d_alcv, 0, ncand * sizeof(double));
    check_gpu_error("alcv copy");

#ifdef TIMINGS
    /* finish timing memory copies */
    gettimeofday(&t2, 0);
    double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
    printf("GPU allocate/copy-in  time: %4.3fs\n", time); 
#endif

    /* run the kernel */
    int nBlocks = ncand; 
    dim3 dimBlock(n, 1, 1);
    dim3 dimGrid(nBlocks, 1, 1);

#ifdef TIMINGS
    /* begin timing of GPU calculation */
    gettimeofday(&t1, 0);
#endif

    /* begin GPU calculation */
    int sh_size = (3*n + nref + n*m)*sizeof(double);
    alc_kernel<<<dimGrid,dimBlock,sh_size>>>(n, m, d_X, d_Ki, d, g, phi, ncand, 
          d_Xcand, nref, d_Xref, d_k, d_alcv);
    hipDeviceSynchronize();

    /* check for errors */
    check_gpu_error("alc_kernel");

#ifdef TIMINGS
    /* finish calculation timing */
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
    printf("GPU compute time: %4.3fs\n", time); 

    /* begin timing of output copies */
    gettimeofday(&t1, 0);
#endif

    /* copy from GPU */
    hipMemcpy(alcv, d_alcv, ncand * sizeof(double), hipMemcpyDeviceToHost);

    /* clean up CUDA */
    hipFree(d_X);
    hipFree(d_alcv);
    hipFree(d_Ki);
    hipFree(d_Xref);
    hipFree(d_Xcand);
    hipFree(d_k);


#ifdef TIMINGS
    /* end timing of output copies */
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000000.0;
    printf("GPU free/copy-out time: %4.2fs\n", time); 
#endif
  }
}
